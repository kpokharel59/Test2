#include "hip/hip_runtime.h"
 #include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define BLOCK_SIZE 16
#define GRID_SIZE 160
#define SIZE BLOCK_SIZE*BLOCK_SIZE*GRID_SIZE*GRID_SIZE

texture<float,1,hipReadModeElementType> tex_1DA;
texture<float,2,hipReadModeElementType> tex_A;
surface<void,2> surf;

void checkresult(float *ref, float *in, float *out, float *mul, int width){
	
	for(int i = 0 ; i < GRID_SIZE; i++){
		for(int j = 0; j < GRID_SIZE; j++){
			float sum = 0.0f;
			int start = j * BLOCK_SIZE * width + i * BLOCK_SIZE;
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					sum += in[start + ii * width + jj] * mul[jj];
				}
			}
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					if(jj % 2 == 0 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 2.0 * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 1)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = (-1.0) * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 0.0f;
				}
			}
		}
	}

	for(int i = 0; i < SIZE; i++){
		if(abs(ref[i]-out[i]) > 1.e-6){
			printf("results checking failed at %d ref %f out %f\n", i, ref[i], out[i]);
			return;
		}
	}
	printf("results checking passed!\n");
}

__global__ void norm(float *in, float *out, float *mul, int width){
	unsigned int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	unsigned int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;

	if(tx >= width || ty >= SIZE/width) return;
	int start = blockIdx.x * BLOCK_SIZE * width + blockIdx.y * BLOCK_SIZE;
	float sum = 0.0f;

	for(int i = 0; i < BLOCK_SIZE; i++){
		for(int j = 0; j < BLOCK_SIZE; j++){
			sum += in[start + i * width + j] * mul[j];
		}
	}
	
	//surf2Dwrite((2.0 * in[tx * width + ty]/sum),surf,ty*sizeof(float),tx);
	if(tx % 2 == 0 && ty % 2 == 0)
		surf2Dwrite((2.0 * in[tx * width + ty]/sum),surf,tx*sizeof(float),ty);
		//out[tx * width + ty] = 2.0 * in[tx * width + ty]/sum;
	else if(tx % 2 == 1 && ty % 2 == 0)
		surf2Dwrite((in[tx * width + ty]/sum),surf,tx*sizeof(float),ty);
		//out[tx * width + ty] = in[tx * width + ty]/sum;
	else if(tx % 2 == 1 && ty % 2 == 1)
		surf2Dwrite(((-1.0) * in[tx * width + ty]/sum),surf,tx*sizeof(float),ty);
		//out[tx * width + ty] = (-1.0) * in[tx * width + ty]/sum;
	else
		surf2Dwrite((0.0f),surf,tx*sizeof(float),ty);
		//out[tx * width + ty] = 0.0f;

}



int main(){
	float *hA_in = (float *)malloc(SIZE * sizeof(float));
	float *hA_out = (float *)malloc(SIZE * sizeof(float));
	float *hB_in = (float *)malloc(BLOCK_SIZE * sizeof(float));
	float *ref = (float *)malloc(SIZE * sizeof(float));
	float *dA_in, *dA_out, *dB_in;

	srand(2016);

	//
	const unsigned int trans_size= GRID_SIZE * BLOCK_SIZE;
	 
	for(int i = 0; i < SIZE; i++){
		hA_in[i] = (float)rand()/(float)RAND_MAX;
	}
	for(int i = 0; i < BLOCK_SIZE; i++){
		hB_in[i] = (float)rand()/(float)RAND_MAX;
	}

	hipMalloc((void **)&dA_in, SIZE * sizeof(float));
	hipMalloc((void **)&dA_out, SIZE * sizeof(float));
	hipMalloc((void **)&dB_in, BLOCK_SIZE * sizeof(float));

	hipMemcpy(dA_in, hA_in, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB_in, hB_in, BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
	
	//////////
	 hipChannelFormatDesc channelDescA =  hipCreateChannelDesc<float>();
	hipChannelFormatDesc forB= hipCreateChannelDesc<float>();//hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	hipArray* A_Array;
	hipArray* B_Array;
	hipMallocArray(&A_Array, &channelDescA, trans_size,trans_size);
	hipMallocArray(&B_Array, &forB,trans_size,trans_size,hipArraySurfaceLoadStore);
	hipMemcpyToArray(A_Array, 0, 0, hA_in, SIZE,
                      hipMemcpyHostToDevice);
	tex_A.addressMode[0] = hipAddressModeWrap;
    tex_A.addressMode[1] = hipAddressModeWrap;
    tex_A.filterMode     = hipFilterModePoint;
	hipBindTextureToArray(tex_A, A_Array, channelDescA);
	cudaBindSurfaceToArray(surf,B_Array,forB);
	hipBindTexture(0,tex_1DA,dA_in,SIZE);
	//////////
	
	
	struct timespec start, end;	
	dim3 grid(GRID_SIZE, GRID_SIZE, 1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &start);

	norm<<<grid, block>>>(dA_in, dA_out, dB_in, BLOCK_SIZE * GRID_SIZE);

	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &end);

	printf("kernel time %fs\n", end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec)/1.e9);
	//hipMemcpy(hA_out, dA_out, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpyFromArray(hA_out,B_Array,0,0,SIZE,hipMemcpyDeviceToHost);
	checkresult(ref, hA_in, hA_out, hB_in, BLOCK_SIZE * GRID_SIZE);

}
